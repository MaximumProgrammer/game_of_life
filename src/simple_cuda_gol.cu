#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <fstream>
#include <sstream>
#include <stdio.h>

/* #include <cuda_gl_interop.h> */

#include "simple_cuda_gol.cuh"

#define MAXBLOCKS 65535


const GLfloat SimpleCudaGoL::left = 0.0;
const GLfloat SimpleCudaGoL::right = 1.0;
const GLfloat SimpleCudaGoL::bottom = 0.0;
const GLfloat SimpleCudaGoL::top = 1.0;
const GLint SimpleCudaGoL::FPS = 25;
GLfloat SimpleCudaGoL::zoomFactor = 1;
GLfloat SimpleCudaGoL::deltaX = 0.0f;
GLfloat SimpleCudaGoL::deltaY = 0.0f;
GLint SimpleCudaGoL::windowWidth  = 600;
GLint SimpleCudaGoL::windowHeight = 600 ;
SimpleCudaGoL* SimpleCudaGoL::ptr = NULL;



SimpleCudaGoL::SimpleCudaGoL(int N)
{
  // Seed the random number Generator
  srand(time(NULL));

  ptr = this;
  // Set the dimensions of the grid.
  width_ = N;
  height_ = N;

  initDisplay();

  currentGrid_ = new bool[width_ * height_];
  if ( currentGrid_ == NULL )
  {
    std::cout << "Could not allocate memory for the current Grid!" <<
      std::endl;
    std::exit(-1);
  }
  nextGrid_ = new bool[width_ * height_];
  if ( nextGrid_ == NULL )
  {
    std::cout << "Could not allocate memory for the next generation Grid!"
      << std::endl;
    std::exit(-1);
  }

  colorArray_ = new color[width_ * height_];
  if (colorArray_ == NULL)
  {
    std::cout << "Could not allocate memory for the color Array!"
      << std::endl;
    std::exit(-1);
  }

  for(int i = 0; i < height_; i++)
  {
    for(int j = 0; j < width_; j++)
    {
      currentGrid_[i * width_ + j] = ( (float)rand() / (float)RAND_MAX )
        < THRESHOLD;
      colorArray_[i * width_ + j] = color(0, 0, 0);
      //currentGrid_[ i * N + j] = BEACON_2[ i * N + j];
    }
  }
}

SimpleCudaGoL::SimpleCudaGoL(std::string fileName): genCnt_(0)
{
  bool parseFlag = parseConfigFile(fileName);
  if (!parseFlag)
  {
    std::cout << "Game of Life will not begin!Exiting now!" << std::endl;
    std::exit(-1);
  }

  currentGrid_ = new bool[width_ * height_];
  if ( currentGrid_ == NULL )
  {
    std::cout << "Could not allocate memory for the current Grid!" <<
      std::endl;
    std::exit(-1);
  }
  nextGrid_ = new bool[width_ * height_];
  if ( nextGrid_ == NULL )
  {
    std::cout << "Could not allocate memory for the next generation Grid!"
      << std::endl;
    std::exit(-1);
  }

  colorArray_ = new color[width_ * height_];
  if (colorArray_ == NULL)
  {
    std::cout << "Could not allocate memory for the color Array!"
      << std::endl;
    std::exit(-1);
  }

  // If the specified input file name is the "random" keyword
  // then create a random initial grid.
  if (inputFileName_.compare("random") == 0)
  {
    srand(time(NULL));
    for(int i = 0; i < height_; i++)
    {
      for(int j = 0; j < width_; j++)
      {
        currentGrid_[i * width_ + j] = ( (float)rand() / (float)RAND_MAX )
          < THRESHOLD;
      }
    }
  }
  else
    // Parse the grid from the file
    utilities::read_from_file(currentGrid_, inputFileName_, width_);

  std::cout << "Successfully created the initial grid!" << std::endl;

  if (displayFlag_)
    initDisplay();

  for(int i = 0; i < height_; i++)
  {
    for(int j = 0; j < width_; j++)
    {
      colorArray_[i * width_ + j] = color(0, 0, 0);
    }
  }
  ptr = this;

  std::cout << "Created Game of Life Object!" << std::endl;
  return;
}

/**
 * @brief Initialize all the functions used for displaying the grid.
 */
void SimpleCudaGoL::initDisplay(void)
{
  glutInitWindowSize(windowWidth , windowHeight);
  glutInitWindowPosition(0, 0);
  windowId_ = glutCreateWindow("Game of Life");
  glClearColor(0, 0, 0, 0);

  glutDisplayFunc(SimpleCudaGoL::display);
  glutIdleFunc(SimpleCudaGoL::getNextGenerationWrapper);
  glutKeyboardFunc(SimpleCudaGoL::keyBoardCallBack);
  glutSpecialFunc(SimpleCudaGoL::arrowKeyCallback);
}

void SimpleCudaGoL::reshape(int w , int h)
{
  windowWidth = w;
  windowHeight = h;

  glViewport(0, 0, windowWidth, windowHeight);

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glOrtho(0.f, windowWidth , windowHeight, 0.f, 0.f , 1.f);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  glutPostRedisplay();
}


bool SimpleCudaGoL::parseConfigFile(std::string fileName)
{
  std::cout << "Parsing Input File!" << std::endl;

  std::ifstream configFile;
  configFile.open(fileName.c_str());
  if (!configFile.is_open())
  {
    std::cout << "Could not open the configuration file!" << std::endl;
    return false;
  }

  std::string line;
  std::getline(configFile, line);
  while (configFile)
  {
    // Ignore tab lines, carriage return, newline and the # characters
    if ((line.find_first_not_of(" \t\r\n") != std::string::npos) && (line[0] != '#'))
    {
      std::stringstream ss(line);
      std::string command;
      ss >> command;

      // Get the size of the Game Of Life Grid.
      if (command.compare("width") == 0)
      {
        ss >> width_;
        if (ss.fail())
        {
          std::cout << "Could not read the width of the grid!" << std::endl;
          return false;
        }
      }
      else if (command.compare("height") == 0)
      {
        ss >> height_;
        if (ss.fail())
        {
          std::cout << "Could not read the height of the grid!" << std::endl;
          return false;
        }
      }
      // Get the name of the file where the board is stored.
      else if (command.compare("boardFileName") == 0)
      {
        ss >> inputFileName_;
        if (ss.fail())
        {
          std::cout << "Could not read the name of the file containing the board!" << std::endl;
          return false;
        }
      }
      // Parse the option specifying whether to display the game or not.
      else if (command.compare("display") == 0)
      {
        std::string displayStr;
        ss >> displayStr;
        if (ss.fail())
        {
          std::cout << "Could not read the value for the display flag, setting it to false!" << std::endl;
          displayFlag_ = false;
        }
        else
          displayFlag_ = displayStr.compare("true") == 0;
      }
      // Get the maximum number of generations for the game.
      else if (command.compare("generationNumber") == 0)
      {
        ss >> maxGenerationNumber_;
        if (ss.fail())
        {
          std::cout << "Could not parse the max number of generations!" << std::endl;
          return false;
        }
      }
      // If present, read the name of the file where the
      else if (command.compare("outputFile") == 0)
      {
        ss >> outputFileName_;
        if (ss.fail())
          std::cout << "Could not read the name of the output grid file!"
            << " The result will not be saved!"<< std::endl;
      }
    } // End of If clause for invalid characters.

    // Get the next line of the file.
    getline(configFile, line);
  } // End of While loop.

  std::cout << "Finished Reading the configuration file!" << std::endl;
  return true;
}



void SimpleCudaGoL::display()
{
  // Clear the buffer.
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  // Calculate the size of each cell in each direction.
  GLfloat xSize = zoomFactor * (right - left) / ptr->width_;
  GLfloat ySize = zoomFactor * (top - bottom) / ptr->height_;
  GLint width = ptr->width_;
  GLint height = ptr->height_;

  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  glTranslatef(deltaX, deltaY, 0.0f);

  glMatrixMode(GL_MODELVIEW);
  // Load the identity transformation matrix.
  glLoadIdentity();
  // Define the scale transformation so as to properly view the grid.
  glScalef(xSize, ySize, 1.0f);
  // Apply a translation transformation so as to place the center of the grid
  // on the center of the window and move it when the user moves it using the
  // keyboard arrow keys.
  glTranslatef(-width / 2.0f, height / 2.0f, 0.0f);

  glBegin(GL_QUADS);

  // TO DO : Replace vertex drawing with faster method(vertex array or VBO) for
  // faster rendering.
  for (GLint y = 0; y < height; ++y)
  {
    for (GLint x = 0; x < width ; ++x)
    {
      int index = y * width + x;
      // At this point, the nextGrid array contains the information abou the last generation
      // of the game.
      // If the current cell was dead and is revived
      if (ptr->currentGrid_[index] && !ptr->nextGrid_[index])
        ptr->colorArray_[index] = color(0, 128, 0);

      // If the cell was alive and died.
      if (!ptr->currentGrid_[index] && ptr->nextGrid_[index])
        ptr->colorArray_[index] = color(128, 0, 0);
      else
        ptr->colorArray_[index].red > 0 ? ptr->colorArray_[index].red-- : 0;

      // If the cell remains alive.
      if (ptr->currentGrid_[index])
        ptr->colorArray_[index].green >= 255 ? ptr->colorArray_[index].green = 255:
          ptr->colorArray_[index].green++;

      // Update the current color.
      glColor3ub(ptr->colorArray_[index].red, ptr->colorArray_[index].green,
          ptr->colorArray_[index].blue);
      // Draw the vertex.
      glVertex2f(x, -y - 1);
      glVertex2f(x + 1, -y - 1);
      glVertex2f(x + 1, -y);
      glVertex2f(x, -y);
    }
  }
  glEnd();
  glFlush();
  glutSwapBuffers();

}

void SimpleCudaGoL::keyBoardCallBack(unsigned char key, int x, int y)
{
  // TO DO : Add Arrow Key support
  switch(key)
  {
    case '+':
      zoomFactor += 0.1f;
      break;
    case 'r':
    case 'R':
      zoomFactor = 1.0f;
      deltaX = 0.0f;
      deltaY = 0.0f;
      break;
    case '-':
      zoomFactor -= 0.1f;
      break;
    // If the Escape key was pressed then free the allocated resources and std::exit.
    case char(27):
      ptr->terminate();
      break;
    default:
      break;
  }
  if (zoomFactor < 0)
    zoomFactor = 0.2;
  glutPostRedisplay();
}

void SimpleCudaGoL::arrowKeyCallback(int key, int x, int y)
{
  switch (key)
  {
    case GLUT_KEY_LEFT: //left
      deltaX -= 0.005f;
      break;
    case GLUT_KEY_UP: //up
      deltaY += 0.005f;
      break;
    case GLUT_KEY_RIGHT: //right
      deltaX += 0.005f;
      break;
    case GLUT_KEY_DOWN: //down
      deltaY -= 0.005f;
      break;
    default:
      break;
  }
}


void SimpleCudaGoL::terminate()
{
  std::cout << "Terminating Game of Life!" << std::endl;
  delete[] currentGrid_;
  delete[] nextGrid_;
  glutDestroyWindow(windowId_);
  std::exit(0);
  return;
}

void SimpleCudaGoL::play()
{
  std::cout << "Starting to play!" << std::endl;
  const size_t arraySize = width_ * height_;

  bool* currentGridDevice;
  bool* nextGridDevice;

  bool* startingGrid = new bool[width_ * height_];
  memcpy(startingGrid, currentGrid_, arraySize * sizeof(bool));

  bool* tempArray = new bool[width_ * height_];

  hipMalloc((void**) &currentGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << "Unable to allocate Device Memory!" << std::endl;
    terminate();
  }

  dim3 threadNum(16, 16);
  dim3 blocks(width_ / threadNum.x + 1, height_ / threadNum.y + 1);

  // TO DO: Add CUDA - OpenGL Interoperability
  /* if (!displayFlag_) */
  /* { */
  struct timeval startTime, endTime;
  gettimeofday(&startTime, NULL);

  for (int i = 0; i < maxGenerationNumber_; ++i)
  {
    getNextGeneration();
  }
  gettimeofday(&endTime, NULL);
  double serialExecTime = (double)((endTime.tv_usec - startTime.tv_usec)
      /1.0e6 + endTime.tv_sec - startTime.tv_sec);
  std::cout << "Serial Execution Time is = " << serialExecTime << std::endl;

  memcpy(currentGrid_, startingGrid, arraySize * sizeof(bool));

  hipEvent_t startTimeDevice, endTimeDevice;
  /* gettimeofday(&startTime, NULL); */
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, startingGrid, arraySize * sizeof(bool), hipMemcpyHostToDevice);
  for (int i = 0; i < maxGenerationNumber_; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, width_);
    cudaCheckErrors("Exec Error");
    SWAP(currentGridDevice, nextGridDevice);
    /* getNextGeneration(); */
    /* hipMemcpy(tempArray, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost); */

    /* hipMemcpy(currentGrid_, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost); */
    /* cudaCheckErrors("Mem Copy Error"); */
    /* compareArray(tempArray, currentGrid_, width_); */
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(currentGrid_, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << "GPU Execution Time is = " << time / 1000.0f  << std::endl;
  std::cout << "Serial Time / GPU Time = " << static_cast<int>(serialExecTime / (time / 1000.0f)) << std::endl;


  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] startingGrid;

  /* } */
  /* else */
  /* { */
    /* gettimeofday(&startTime, NULL); */
    /* glutMainLoop(); */
  /* } */

  std::cout << "Finished playing the game of Life!" << std::endl;
}

void SimpleCudaGoL::getNextGenerationWrapper()
{
  if (ptr == NULL)
  {
    std::cout << "The pointer to the function has not been initialized!"
      << std::endl;
    std::exit(-1);
  }
  if (ptr->genCnt_ > ptr->maxGenerationNumber_)
    ptr->terminate();

  // gettimeofday(&ptr->startTime, NULL);
  /* ptr->getNextGeneration(); */
  // gettimeofday(&ptr->endTime, NULL);

  // double nextGenTime = (double)((ptr->endTime.tv_usec - ptr->startTime.tv_usec)
      // /1.0e6 + ptr->endTime.tv_sec - ptr->startTime.tv_sec);

  // std::cout << std::endl << "Next Gen Time = " << nextGenTime << std::endl;

  glutPostRedisplay();
  return;
}

void SimpleCudaGoL::getNextGeneration()
{
  int width = width_;
  int height = height_;
  for (int y = 0; y < height; ++y)
  {
    size_t up = ( (y + height - 1) % height) * width;
    size_t center = y * width;
    size_t down = ((y + 1) % height) * width;
    for (int x = 0; x < width; ++x)
    {
      size_t left = (x + width - 1) % width;
      size_t right = (x + 1) % width;

      int livingNeighbors = calcNeighbors(x , left, right, center, up, down);
      nextGrid_[center + x] = livingNeighbors == 3 ||
        (livingNeighbors == 2 && currentGrid_[x + center]) ? 1 : 0;
    }
  }
  // Set the next generation grid as the current on for the next iteration
  // of the algorithm.
  // TO DO : Make into MACRO
  std::swap(currentGrid_, nextGrid_);
  return;
}

int SimpleCudaGoL::calcNeighbors(int x, int left, int right, int center,
    int up, int down)
{
  return currentGrid_[left + up] + currentGrid_[x + up]
      + currentGrid_[right + up] + currentGrid_[left + center]
      + currentGrid_[right + center] + currentGrid_[left + down]
      + currentGrid_[x + down] + currentGrid_[right + down];
}


__global__ void simpleNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * N + col;
  if (index > N * N)
    return;

  int x = index % N;
  int y = (index - x) / N;
  size_t up = ( (y + N - 1) % N) * N;
  size_t center = y * N;
  size_t down = ((y + 1) % N) * N;
  size_t left = (x + N - 1) % N;
  size_t right = (x + 1) % N;

  int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
  nextGrid[center + x] = livingNeighbors == 3 ||
    (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;

  return;
}

__device__ int calcNeighborsKernel(bool* currentGrid, int x, int left, int right, int center,
    int up, int down)
{
  return currentGrid[left + up] + currentGrid[x + up]
      + currentGrid[right + up] + currentGrid[left + center]
      + currentGrid[right + center] + currentGrid[left + down]
      + currentGrid[x + down] + currentGrid[right + down];
}
