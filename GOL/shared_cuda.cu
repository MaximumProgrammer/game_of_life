#include "hip/hip_runtime.h"
#include <iostream>

#include "shared_cuda.cuh"
#include "utilities.cuh"

#define TILE_SIZE 16
#define TILE_SIZE_X 16
#define TILE_SIZE_Y 16
#define CELLS_PER_THR 2
#define MAXBLOCKS 512


void singleCellSharedMem(bool* startingGrid, int N, int maxGen){
  std::string prefix("[Shared Memory Single Cell per Thread]: ");

  int GhostN = N + 2;
  bool* initialGameGrid = new bool[GhostN * GhostN];
  if (initialGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the initial grid array!" << std::endl;
    return;
  }

  utilities::generate_ghost_table(startingGrid, initialGameGrid, N);

  bool* finalGameGrid = new bool[GhostN * GhostN];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t currentGridPitch;
  size_t nextGridPitch;

  hipMallocPitch((void**) &currentGridDevice, &currentGridPitch, GhostN * sizeof(bool), GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMallocPitch((void**) &nextGridDevice, &nextGridPitch, GhostN * sizeof(bool), GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  // Execute the second version of the many cells per thread gpu implementation.
  dim3 threadNum(TILE_SIZE_X, TILE_SIZE_Y);
  dim3 blocks(std::min(N / (threadNum.x) + 1, (unsigned int)MAXBLOCKS),
      std::min(N / (threadNum.y) + 1, (unsigned int)MAXBLOCKS));

  dim3 ghostMatThreads(16, 1);
  dim3 ghostGridRowsSize(N / ghostMatThreads.x + 1, 1);//It will not copy the corners
  dim3 ghostGridColSize(N / ghostMatThreads.x + 1, 1);//It coppies corners tooo

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  // Copy the initial grid to the device.
  hipMemcpy2D(currentGridDevice, currentGridPitch, initialGameGrid, GhostN * sizeof(bool), GhostN * sizeof(bool)
      , GhostN, hipMemcpyHostToDevice);
  /* cudaCheckErrors("Initial MemCpy 2d"); */
  for (int i = 0; i < maxGen; ++i)
  {
    // Update the ghost elements of the Array
    utilities::updateGhostRows<<< ghostGridRowsSize, ghostMatThreads>>>(currentGridDevice, GhostN, currentGridPitch);
    utilities::updateGhostCols<<< ghostGridColSize, ghostMatThreads>>>(currentGridDevice, GhostN, currentGridPitch);
    utilities::updateGhostCorners<<< 1, 1 >>>(currentGridDevice, GhostN, currentGridPitch);
    singleCellSharedMemKernel<<< blocks, threadNum >>>(currentGridDevice, nextGridDevice, N,
        currentGridPitch, nextGridPitch);
    hipDeviceSynchronize();
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy2D(finalGameGrid, GhostN * sizeof(bool), currentGridDevice, currentGridPitch, GhostN * sizeof(bool),
      GhostN, hipMemcpyDeviceToHost);
  /* cudaCheckErrors("Final MemCpy Error"); */

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);

  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f  << "> seconds" << std::endl;
  utilities::countGhost(finalGameGrid, N, N, prefix);

  // Free device memory.
  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  // Free host memory.
  delete[] finalGameGrid;

  return;
}


void multiCellSharedMemPitch(bool* startingGrid, int N, int maxGen){
  std::string prefix("[Shared Memory Multiple Cells per Thread Pitch]: ");

  int GhostN = N + 2;
  bool* initialGameGrid = new bool[GhostN * GhostN];
  if (initialGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the initial grid array!" << std::endl;
    return;
  }

  utilities::generate_ghost_table(startingGrid, initialGameGrid, N);

  bool* finalGameGrid = new bool[GhostN * GhostN];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t currentGridPitch;
  size_t nextGridPitch;

  hipMallocPitch((void**) &currentGridDevice, &currentGridPitch, GhostN * sizeof(bool), GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMallocPitch((void**) &nextGridDevice, &nextGridPitch, GhostN * sizeof(bool), GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  // Execute the second version of the many cells per thread gpu implementation.
  dim3 threadNum(TILE_SIZE_X, TILE_SIZE_Y);
  dim3 blocks(std::min(GhostN / (threadNum.x * CELLS_PER_THR) + 1, (unsigned int)MAXBLOCKS),
      std::min(GhostN / (threadNum.y * CELLS_PER_THR) + 1, (unsigned int)MAXBLOCKS));

  dim3 ghostMatThreads(16, 1);
  dim3 ghostGridRowsSize(N / ghostMatThreads.x + 1, 1);//It will not copy the corners
  dim3 ghostGridColSize(N / ghostMatThreads.x + 1, 1);//It coppies corners tooo

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  // Copy the initial grid to the device.
  hipMemcpy2D(currentGridDevice, currentGridPitch, initialGameGrid, GhostN * sizeof(bool), GhostN * sizeof(bool)
      , GhostN, hipMemcpyHostToDevice);
  /* cudaCheckErrors("Initial MemCpy 2d"); */
  for (int i = 0; i < maxGen; ++i)
  {
    // Update the ghost elements of the Array
    utilities::updateGhostRows<<< ghostGridRowsSize, ghostMatThreads>>>(currentGridDevice, GhostN, currentGridPitch);
    utilities::updateGhostCols<<< ghostGridColSize, ghostMatThreads>>>(currentGridDevice, GhostN, currentGridPitch);
    utilities::updateGhostCorners<<< 1, 1 >>>(currentGridDevice, GhostN, currentGridPitch);
    multiCellSharedMemPitchKernel<<< blocks, threadNum >>>(currentGridDevice, nextGridDevice, N,
        currentGridPitch, nextGridPitch);
    hipDeviceSynchronize();
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy2D(finalGameGrid, GhostN * sizeof(bool), currentGridDevice, currentGridPitch, GhostN * sizeof(bool),
      GhostN, hipMemcpyDeviceToHost);
  /* cudaCheckErrors("Final MemCpy Error"); */

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);

  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f  << "> seconds" << std::endl;
  utilities::countGhost(finalGameGrid, N, N, prefix);

  // Free device memory.
  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  // Free host memory.
  delete[] finalGameGrid;

  return;
}

void multiCellSharedMem(bool* startingGrid, int N, int maxGen){
  std::string prefix("[Shared Memory Multiple Cells per Thread ]: ");
  int GhostN = N + 2;

  bool* initialGameGrid = new bool[(GhostN) * (GhostN)];
  if (initialGameGrid == NULL){
    std::cout << prefix << "Could not allocate memory for the initial grid array!" << std::endl;
    return;
  }

  bool* finalGameGrid = new bool[(GhostN) * (GhostN)];
  if (finalGameGrid == NULL){
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  utilities::generate_ghost_table(startingGrid, initialGameGrid, N);
  /* utilities::print(initialGameGrid, N + 2); */
  bool* currentGridDevice;
  bool* nextGridDevice;

  hipMalloc((void**) &currentGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL){
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(TILE_SIZE, TILE_SIZE);
  //imperfect division creates problems(we have to use if)
  /* dim3 blocks(GhostN/(threadNum.x * CELLS_PER_THR) + 1, GhostN/( threadNum.y * CELLS_PER_THR) + 1);//CREATE MACRO CALLED CEIL */

  /* dim3 blocks(N / (threadNum.x * CELLS_PER_THR) + 1, */
      /* N / (threadNum.y * CELLS_PER_THR) + 1);//CREATE MACRO CALLED CEIL */

  dim3 blocks(std::min(
        (N  + (threadNum.x * CELLS_PER_THR) -1) / (threadNum.x * CELLS_PER_THR), (unsigned int)MAXBLOCKS),
      std::min(
        (N +(threadNum.y * CELLS_PER_THR) -1)/ (threadNum.y * CELLS_PER_THR) , (unsigned int)MAXBLOCKS));

  dim3 ghostMatThreads(16, 1);
  dim3 ghostGridRowsSize(N / ghostMatThreads.x + 1, 1);
  dim3 ghostGridColSize(N / ghostMatThreads.x + 1, 1);
  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, initialGameGrid, GhostN * GhostN, hipMemcpyHostToDevice);

  for (int i = 0; i < maxGen; ++i)
  {
    utilities::updateGhostRows<<< ghostGridRowsSize, ghostMatThreads>>>(currentGridDevice, GhostN, GhostN);
    utilities::updateGhostCols<<< ghostGridColSize, ghostMatThreads>>>(currentGridDevice, GhostN, GhostN);
    utilities::updateGhostCorners<<< 1, 1 >>>(currentGridDevice, GhostN, GhostN);
    multiCellSharedMemKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice,  N);
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(finalGameGrid, currentGridDevice, GhostN *GhostN , hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;

  utilities::countGhost(finalGameGrid, N, N, prefix);
  /* utilities::print(finalGameGrid, N + 2); */
  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;
  return;
}

__global__ void singleCellSharedMemKernel(bool* currentGrid, bool* nextGrid, int N, size_t currentGridPitch,
    size_t nextGridPitch)
{
  size_t startPoint = __umul24(blockIdx.y , blockDim.y) * currentGridPitch +
    __umul24(blockIdx.x, blockDim.x);

  size_t row = __umul24(blockIdx.y, blockDim.y) + threadIdx.y + 1;
  size_t col = __umul24(blockIdx.x, blockDim.x) + threadIdx.x + 1;

  int i = threadIdx.y;
  int j = threadIdx.x;

  __shared__ bool localGrid[TILE_SIZE_Y + 2][TILE_SIZE_X + 2];



  int linIndex = __umul24(i, TILE_SIZE_X) + j;
  int jj = linIndex % (TILE_SIZE_X + 2);
  int ii = (linIndex - jj) / (TILE_SIZE_X + 2);
  int I = ii * currentGridPitch + jj;

  localGrid[ii][jj] = currentGrid[startPoint + I];

  int linIndex2 = __umul24(TILE_SIZE_Y, TILE_SIZE_X) + linIndex;
  int jj2 = linIndex2 % (TILE_SIZE_X + 2);
  int ii2 = (linIndex2 - jj2) / (TILE_SIZE_X + 2);

  int I2 = ii2 * currentGridPitch + jj2;

  if ((jj2 < TILE_SIZE_X + 2) && (ii2 < TILE_SIZE_Y + 2) && (I2 < (N+2) * (N+2)))
  {
    localGrid[ii2][jj2] = currentGrid[startPoint + I2];
  }

  __syncthreads();

  if ((row < N + 1) && (col < N + 1))
  {
    i++;
    j++;
    int livingNeighbors = localGrid[i - 1][j - 1] + localGrid[i - 1][j]
      + localGrid[i - 1][j + 1] + localGrid[i][j - 1] + localGrid[i][j + 1]
      + localGrid[i + 1][j - 1] + localGrid[i + 1][j] + localGrid[i + 1][j + 1];
    nextGrid[row * nextGridPitch + col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && localGrid[i][j]) ? 1 : 0;
  }

  return;
}

__global__ void multiCellSharedMemKernel(bool* currentGrid, bool* nextGrid, int N){

  int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

  /* int xStride = __umul24(blockDim.x, gridDim.x); */
  /* int yStride = __umul24(blockDim.y, gridDim.y); */

  int xStride = blockDim.x * gridDim.x;
  int yStride = blockDim.y * gridDim.y;

  int i = threadIdx.y;
  int j = threadIdx.x;

  int threadRowIndex = i + 1;
  int threadColIndex = j + 1;

  /* __shared__ bool localGrid[(TILE_SIZE_Y + 2) * (TILE_SIZE_X + 2)]; */
  size_t startPoint = blockIdx.y * blockDim.y * (N + 2) + blockIdx.x * blockDim.x;

  int linIndex = i * TILE_SIZE_X + j;
  int jj = linIndex % (TILE_SIZE_X + 2);
  int ii = (linIndex - jj) / (TILE_SIZE_X + 2);
  int I = ii * (N + 2) + jj;

  int linIndex2 = TILE_SIZE_X * TILE_SIZE_Y + linIndex;
  int jj2 = linIndex2 % (TILE_SIZE_X + 2);
  int ii2 = (linIndex2 - jj2) / (TILE_SIZE_X + 2);
  int I2 = ii2 * (N + 2) + jj2;

  __shared__ bool localGrid[TILE_SIZE_Y + 2][TILE_SIZE_X + 2];

  for (int m = 0; m < CELLS_PER_THR; m++)
  {
    size_t row = yIndex + m * yStride + 1;
    /* size_t row = yIndex + __umul24(m, yStride) + 1; */
    /* for (int n = xIndex; n < N + TILE_SIZE; n += xStride) */
    size_t nextRow = row * (N + 2);

    for (int n = 0; n < CELLS_PER_THR; n++)
    {
      size_t col = xIndex + n * xStride + 1;
      /* size_t col = xIndex + __umul24(n, xStride) + 1; */
      /* startPoint = __umul24(m - i, N + 2) + n - j; */
      /* startPoint = (m - i) * (N + 2) + n - j; */
      startPoint = (row - i - 1) * (N + 2) + col - j - 1;

      localGrid[ii][jj] = currentGrid[startPoint + I];

      if ((jj2 < TILE_SIZE_X + 2) && (ii2 < TILE_SIZE_Y + 2) && (I2 < (N+2) * (N+2)))
        localGrid[ii2][jj2] = currentGrid[startPoint + I2];

      __syncthreads();

      if ((row < N + 1) && (col < N + 1))
      {
        int livingNeighbors = localGrid[threadRowIndex - 1][threadColIndex - 1]
          + localGrid[threadRowIndex - 1][threadColIndex]
          + localGrid[threadRowIndex - 1][threadColIndex + 1]
          + localGrid[threadRowIndex][threadColIndex - 1]
          + localGrid[threadRowIndex][threadColIndex + 1]
          + localGrid[threadRowIndex + 1][threadColIndex - 1]
          + localGrid[threadRowIndex + 1][threadColIndex]
          + localGrid[threadRowIndex + 1][threadColIndex + 1];
        nextGrid[nextRow + col] = livingNeighbors == 3 ||
          (livingNeighbors == 2 && localGrid[threadRowIndex][threadColIndex]) ? 1 : 0;

      }
      __syncthreads();
    }
  }

  return;
}


__global__ void multiCellSharedMemPitchKernel(bool* currentGrid, bool* nextGrid, int N, size_t currentGridPitch,
    size_t nextGridPitch){

  //Copy the neccesary cells to the shared Memory
  int xIndex = blockIdx.x * blockDim.x + threadIdx.x +1;
  int yIndex = blockIdx.y * blockDim.y + threadIdx.y +1;

  int xStride = __umul24(blockDim.x, gridDim.x);
  int yStride = __umul24(blockDim.y, gridDim.y);


  int threadRowIndex = threadIdx.y + 1;
  int threadColIndex = threadIdx.x + 1;

  __shared__ bool localGrid[TILE_SIZE_Y + 2][TILE_SIZE_X + 2];

  for (int i = yIndex; i <= N + 1; i += yStride)
  {
    size_t y = __umul24(i, currentGridPitch);
    size_t yNext = __umul24(i, nextGridPitch);
    size_t up = __umul24(i - 1, currentGridPitch);
    size_t down = __umul24(i + 1, currentGridPitch);

    for (int j = xIndex; j <= N + 1; j += xStride)
    {
      size_t left = j - 1;
      size_t right = j + 1;

      localGrid[threadRowIndex][threadColIndex] = currentGrid[y + j];

      if (threadRowIndex == 1)
      {
        localGrid[threadRowIndex - 1][threadColIndex] = currentGrid[up + j];
        localGrid[threadColIndex][threadRowIndex - 1] =
          currentGrid[(i - threadIdx.y + threadIdx.x) * currentGridPitch - 1];
      }
        /* localGrid[threadRowIndex - 1][threadColIndex] = currentGrid[up + j]; */
      /* if (threadColIndex == 1) */
        /* localGrid[threadRowIndex][threadColIndex - 1] = currentGrid[y  + left]; */
      if (threadRowIndex == TILE_SIZE_Y)
        localGrid[threadRowIndex + 1][threadColIndex] = currentGrid[down + j];
      if (threadColIndex == TILE_SIZE_X)
        localGrid[threadRowIndex][threadColIndex + 1] = currentGrid[y + right];

      if (threadRowIndex == 1 && threadColIndex == 1)
        localGrid[threadRowIndex - 1][threadColIndex - 1] = currentGrid[up + left];
      if (threadRowIndex == 1 && threadColIndex == TILE_SIZE_X)
        localGrid[threadRowIndex - 1][threadColIndex + 1] = currentGrid[up + right];
      if (threadRowIndex == TILE_SIZE_Y && threadColIndex == 1)
        localGrid[threadRowIndex + 1][threadColIndex - 1] = currentGrid[down + left];
      if (threadRowIndex == TILE_SIZE_Y && threadColIndex == TILE_SIZE_X)
        localGrid[threadRowIndex + 1][threadColIndex + 1] = currentGrid[down + right];

      __syncthreads();

      int livingNeighbors = localGrid[threadRowIndex - 1][threadColIndex - 1]
        + localGrid[threadRowIndex - 1][threadColIndex]
        + localGrid[threadRowIndex - 1][threadColIndex + 1]
        + localGrid[threadRowIndex][threadColIndex - 1]
        + localGrid[threadRowIndex][threadColIndex + 1]
        + localGrid[threadRowIndex + 1][threadColIndex - 1]
        + localGrid[threadRowIndex + 1][threadColIndex]
        + localGrid[threadRowIndex + 1][threadColIndex + 1];
      nextGrid[yNext + j] = livingNeighbors == 3 ||
        (livingNeighbors == 2 && localGrid[threadRowIndex][threadColIndex]) ? 1 : 0;

    }
  }
  return;
}


__device__ int sharedCalcNeighborsKernel(bool* currentGrid, size_t x, size_t left, size_t right, size_t center,
    size_t up, size_t down){
  return currentGrid[left + up] + currentGrid[x + up]
    + currentGrid[right + up] + currentGrid[left + center]
    + currentGrid[right + center] + currentGrid[left + down]
    + currentGrid[x + down] + currentGrid[right + down];
}
