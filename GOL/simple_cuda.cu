#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <iostream>
#include "simple_cuda.cuh"
#define MAXBLOCKS 65535

void simple_cuda(bool** startingGrid, bool** finalGrid, int N, int maxGen)
{
  const size_t arraySize = N* N;

  bool* currentGridDevice;
  bool* nextGridDevice;

  hipMalloc((void**) &currentGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, *startingGrid, arraySize * sizeof(bool), hipMemcpyHostToDevice);
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N);
    cudaCheckErrors("Exec Error");
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(*finalGrid, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << "GPU Execution Time is = " << time / 1000.0f  << std::endl;

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  return;
}

void simpleCudaPitch(bool** startingGrid, bool** finalGrid, int N, int maxGen)
{
  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t pitchStart;
  size_t pitchDest;

  hipMallocPitch((void**) &currentGridDevice, &pitchStart, N, N);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMallocPitch((void**) &nextGridDevice, &pitchDest, N, N);
  cudaCheckErrors("Device memory Allocation Error!");

  std::cout << "Pitch start " << pitchStart << std::endl;
  std::cout << "Pitch dest " << pitchDest << std::endl;

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy2D(currentGridDevice, pitchStart, *startingGrid, N * sizeof(bool), N, N, hipMemcpyHostToDevice);
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernelPitch<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N, pitchStart);
    cudaCheckErrors("Exec Error");
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy2D(*finalGrid, pitchDest, currentGridDevice, pitchStart, N, N, hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << "GPU Execution Time is = " << time / 1000.0f  << std::endl;

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  return;
}

__global__ void simpleNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * N + col;
  if (index > N * N)
    return;

  int x = index % N;
  int y = (index - x) / N;
  size_t up = ( (y + N - 1) % N) * N;
  size_t center = y * N;
  size_t down = ((y + 1) % N) * N;
  size_t left = (x + N - 1) % N;
  size_t right = (x + 1) % N;

  int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
  nextGrid[center + x] = livingNeighbors == 3 ||
    (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;

  return;
}

__global__ void simpleNextGenerationKernelPitch(bool* currentGrid, bool* nextGrid, int N, size_t pitch)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * N + col;
  if (index > N * N)
    return;

  int x = index % N;
  int y = (index - x) / N;
  size_t up = ( (y + N - 1) % N) * N;
  size_t center = y * N;
  size_t down = ((y + 1) % N) * N;
  size_t left = (x + N - 1) % N;
  size_t right = (x + 1) % N;

  int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
  nextGrid[center + x] = livingNeighbors == 3 ||
    (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;

  return;
}

__device__ int calcNeighborsKernel(bool* currentGrid, int x, int left, int right, int center,
    int up, int down)
{
  return currentGrid[left + up] + currentGrid[x + up]
      + currentGrid[right + up] + currentGrid[left + center]
      + currentGrid[right + center] + currentGrid[left + down]
      + currentGrid[x + down] + currentGrid[right + down];
}

