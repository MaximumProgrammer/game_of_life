#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <iostream>
#include "simple_cuda.cuh"
#define MAXBLOCKS 65535

void simpleCuda(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Naive Single Cell per Thread]: ");

  // The host array that will contain the game of life grid after maxGen generations.
  bool* finalGameGrid = new bool[N * N];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the next generation grid array!" << std::endl;
    return;
  }

  // Copy the input data to
  const size_t arraySize = N* N;

  bool* currentGridDevice;
  bool* nextGridDevice;

  hipMalloc((void**) &currentGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, startingGrid, arraySize * sizeof(bool), hipMemcpyHostToDevice);
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N);
    cudaCheckErrors("Exec Error");
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(finalGameGrid, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;
  utilities::count(finalGameGrid, N, N, prefix);

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;

  return;
}

void simpleCudaPitch(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Naive Single Cell per Thread Pitch]: ");

  bool* finalGameGrid = new bool[N * N];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t pitchStart;
  size_t pitchDest;

  hipMallocPitch((void**) &currentGridDevice, &pitchStart, N * sizeof(bool), N);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMallocPitch((void**) &nextGridDevice, &pitchDest, N * sizeof(bool), N);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy2D(currentGridDevice, pitchStart, startingGrid, N * sizeof(bool), N * sizeof(bool)
      , N, hipMemcpyHostToDevice);
  cudaCheckErrors("Initial Memcpy 2D Error");
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernelPitch<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N, pitchStart,
        pitchDest);
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy2D(finalGameGrid, N * sizeof(bool), currentGridDevice, pitchStart, N * sizeof(bool),
      N, hipMemcpyDeviceToHost);
  cudaCheckErrors("Final Memcpy 2D Error");

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;
  utilities::count(finalGameGrid, N, N, prefix);

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;
  return;
}



void simpleCudaGhostPitch(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Ghosts Single Cell per Thread Pitch]: ");
  int GhostN = N + 2;

  bool* initialGameGrid = new bool[(GhostN) * (GhostN)];
  if (initialGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the initial grid array!" << std::endl;
    return;
  }


  bool* finalGameGrid = new bool[(GhostN) * (GhostN)];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  utilities::generate_ghost_table(startingGrid, initialGameGrid, N);
  /* utilities::print(initialGameGrid, N + 2); */
  bool* currentGridDevice;
  bool* nextGridDevice;

  hipMalloc((void**) &currentGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  //imperfect division creates problems(we have to use if)
  dim3 blocks((GhostN) / threadNum.x + 1, (GhostN) / threadNum.y + 1);//CREATE MACRO CALLED CEIL

  dim3 ghostMatThreads(16, 1);
  dim3 ghostGridRowsSize(N / ghostMatThreads.x + 1, 1);//It will not copy the corners
  dim3 ghostGridColSize(N / ghostMatThreads.x + 1, 1);//It coppies corners tooo

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, initialGameGrid, GhostN * GhostN *sizeof(bool), hipMemcpyHostToDevice);

  for (int i = 0; i < maxGen; ++i)
  {
    utilities::updateGhostRows<<< ghostGridRowsSize, ghostMatThreads>>>(currentGridDevice, GhostN,
        GhostN * sizeof(bool));
    utilities::updateGhostCols<<< ghostGridColSize, ghostMatThreads>>>(currentGridDevice, GhostN,
        GhostN * sizeof(bool));
    utilities::updateGhostCorners<<< 1, 1 >>>(currentGridDevice, GhostN, GhostN * sizeof(bool));
    simpleGhostNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice,  N);
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(finalGameGrid, currentGridDevice, GhostN *GhostN * sizeof(bool), hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;

  utilities::countGhost(finalGameGrid, N, N, prefix);
  /* utilities::print(finalGameGrid, N + 2); */
  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;
  return;
}

__global__ void simpleNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * N + col;
  if (col < N && row < N)
  {
    int x = index % N;
    int y = (index - x) / N;
    size_t up = ((y + N - 1) % N) * N;
    size_t center = y * N;
    size_t down = ((y + 1) % N) * N;
    size_t left = (x + N - 1) % N;
    size_t right = (x + 1) % N;

    int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
    nextGrid[center + x] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;
  }
  return;
}

__global__ void simpleNextGenerationKernelPitch(bool* currentGrid, bool* nextGrid, int N,
    size_t currentGridPitch, size_t nextGridPitch)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < N && row < N)
  {
    bool* currentRow = (bool*)(currentGrid + row * currentGridPitch);

    // The row above the current one.
    size_t up = (row + N - 1) % N;
    bool* previousRow = (bool*)(currentGrid + up * currentGridPitch);
    // The row below the current one.
    size_t down = (row + 1) % N;
    bool* nextRow = (bool*)(currentGrid + down * currentGridPitch);
    // Get the index for the left column
    size_t left = (col + N - 1) % N;
    // Get the index of the right column
    size_t right = (col + 1) % N;


    int livingNeighbors = previousRow[left] + previousRow[col] + previousRow[right]
      + currentRow[left] + currentRow[right] + nextRow[left] + nextRow[col] + nextRow[right];

    bool* nextGridRow = (bool*)(nextGrid + row * nextGridPitch);
    nextGridRow[col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentRow[col]) ? 1 : 0;
  }
  return;
}

__global__ void simpleGhostNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
  if ((col < N + 1) && (row < N + 1))
  {
    size_t up = (row - 1) * (N + 2);
    size_t center = row * (N + 2);
    size_t down = (row + 1) * (N + 2);
    size_t left = col - 1;
    size_t right = col + 1;

    int livingNeighbors = calcNeighborsKernel(currentGrid, col, left, right, center, up, down);
    nextGrid[center + col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentGrid[center + col]) ? 1 : 0;
  }
  return;
}

__global__ void simpleGhostNextGenerationKernelPitch(bool* currentGrid, bool* nextGrid, int N,
    size_t currentGridPitch, size_t nextGridPitch)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
  if ((col < N + 1) && (row < N + 1))
  {
    bool* currentRow = (bool*)(currentGrid + row * currentGridPitch);

    /* The row above the current one. */
    size_t up = (row - 1);
    bool* previousRow = (bool*)(currentGrid + up * currentGridPitch);
    /* The row below the current one. */
    size_t down = (row + 1);
    bool* nextRow = (bool*)(currentGrid + down * currentGridPitch);
    /* Get the index for the left column */
    size_t left = (col - 1);
    /* Get the index of the right column */
    size_t right = (col + 1) ;


    int livingNeighbors = previousRow[left] + previousRow[col] + previousRow[right]
      + currentRow[left] + currentRow[right] + nextRow[left] + nextRow[col] + nextRow[right];

    bool* nextGridRow = (bool*)(nextGrid + row * nextGridPitch);
    nextGridRow[col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentRow[col]) ? 1 : 0;
  }
  return;
}


__device__ int calcNeighborsKernel(bool* currentGrid, int x, int left, int right, int center,
    int up, int down)
{
  return currentGrid[left + up] + currentGrid[x + up]
      + currentGrid[right + up] + currentGrid[left + center]
      + currentGrid[right + center] + currentGrid[left + down]
      + currentGrid[x + down] + currentGrid[right + down];
}

