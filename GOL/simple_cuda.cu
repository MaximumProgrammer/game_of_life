#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <iostream>
#include "simple_cuda.cuh"
#define MAXBLOCKS 65535

void simpleCuda(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Naive Single Cell per Thread]: ");

  // The host array that will contain the game of life grid after maxGen generations.
  bool* finalGameGrid = new bool[N * N];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the next generation grid array!" << std::endl;
    return;
  }

  // Copy the input data to
  const size_t arraySize = N* N;

  bool* currentGridDevice;
  bool* nextGridDevice;

  hipMalloc((void**) &currentGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, arraySize);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy(currentGridDevice, startingGrid, arraySize * sizeof(bool), hipMemcpyHostToDevice);
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N);
    cudaCheckErrors("Exec Error");
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy(finalGameGrid, currentGridDevice, arraySize * sizeof(bool), hipMemcpyDeviceToHost);

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;
  utilities::count(finalGameGrid, N, N, prefix);

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;

  return;
}

void simpleCudaPitch(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Naive Single Cell per Thread Pitch]: ");

  bool* finalGameGrid = new bool[N * N];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t pitchStart;
  size_t pitchDest;

  hipMallocPitch((void**) &currentGridDevice, &pitchStart, N * sizeof(bool), N);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMallocPitch((void**) &nextGridDevice, &pitchDest, N * sizeof(bool), N);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  dim3 blocks(N / threadNum.x + 1, N / threadNum.y + 1);

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  hipMemcpy2D(currentGridDevice, pitchStart, startingGrid, N * sizeof(bool), N * sizeof(bool)
      , N, hipMemcpyHostToDevice);
  cudaCheckErrors("Initial Memcpy 2D Error");
  for (int i = 0; i < maxGen; ++i)
  {
    // Copy the Contents of the current and the next grid
    simpleNextGenerationKernelPitch<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, N, pitchStart,
        pitchDest);
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  hipMemcpy2D(finalGameGrid, N * sizeof(bool), currentGridDevice, pitchStart, N * sizeof(bool),
      N, hipMemcpyDeviceToHost);
  cudaCheckErrors("Final Memcpy 2D Error");

  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;
  utilities::count(finalGameGrid, N, N, prefix);

  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;
  return;
}



void simpleCudaGhostPitch(bool* startingGrid, int N, int maxGen)
{
  std::string prefix("[Ghosts Single Cell per Thread Pitch]: ");
  int GhostN = N + 2;

  bool* initialGameGrid = new bool[(GhostN) * (GhostN)];
  if (initialGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the initial grid array!" << std::endl;
    return;
  }


  bool* finalGameGrid = new bool[(GhostN) * (GhostN)];
  if (finalGameGrid == NULL)
  {
    std::cout << prefix << "Could not allocate memory for the final grid array!" << std::endl;
    return;
  }

  utilities::generate_ghost_table(startingGrid, initialGameGrid, N);
  /* utilities::print(initialGameGrid, N + 2); */
  bool* currentGridDevice;
  bool* nextGridDevice;

  size_t pitchStart;
  size_t pitchDest;

  /* hipMallocPitch((void**) &currentGridDevice, &pitchStart, (GhostN) * sizeof(bool), (GhostN)); */
  /* cudaCheckErrors("Device memory Allocation Error!"); */

  /* hipMallocPitch((void**) &nextGridDevice, &pitchDest, (GhostN) * sizeof(bool), (GhostN)); */
  /* cudaCheckErrors("Device memory Allocation Error!"); */
  hipMalloc((void**) &currentGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  hipMalloc((void**) &nextGridDevice, GhostN *GhostN);
  cudaCheckErrors("Device memory Allocation Error!");

  if (currentGridDevice == NULL || nextGridDevice == NULL)
  {
    std::cout << prefix << "Unable to allocate Device Memory!" << std::endl;
    return;
  }

  dim3 threadNum(16, 16);
  //imperfect division creates problems(we have to use if)
  dim3 blocks((GhostN) / threadNum.x + 1, (GhostN) / threadNum.y + 1);//CREATE MACRO CALLED CEIL

  dim3 ghostMatThreads(16, 1);
  dim3 ghostGridRowsSize(N / ghostMatThreads.x + 1, 1);//It will not copy the corners
  dim3 ghostGridColSize(N / ghostMatThreads.x + 1, 1);//It coppies corners tooo

  hipEvent_t startTimeDevice, endTimeDevice;
  hipEventCreate(&startTimeDevice);
  cudaCheckErrors("Event Initialization Error");
  hipEventCreate(&endTimeDevice);
  cudaCheckErrors("Event Initialization Error");

  hipEventRecord(startTimeDevice, 0);
  /* Copy the initial grid to the device. */
  /* hipMemcpy2D(currentGridDevice, pitchStart, initialGameGrid, GhostN * sizeof(bool), GhostN * sizeof(bool) */
      /* , GhostN, hipMemcpyHostToDevice); */
  /* cudaCheckErrors("Initial Memcpy 2D Error"); */
  hipMemcpy(currentGridDevice, initialGameGrid, GhostN * GhostN *sizeof(bool), hipMemcpyHostToDevice);

  for (int i = 0; i < maxGen; ++i)
  {
    utilities::ghostRows<<< ghostGridRowsSize, ghostMatThreads>>>(currentGridDevice, GhostN);
    utilities::ghostCols<<< ghostGridColSize, ghostMatThreads>>>(currentGridDevice, GhostN);
    utilities::ghostCorners<<< 1, 1 >>>(currentGridDevice, GhostN);
    /* simpleGhostNextGenerationKernelPitch<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice, */
        /* N, pitchStart, pitchDest); */
    simpleGhostNextGenerationKernel<<<blocks, threadNum>>>(currentGridDevice, nextGridDevice,  N);
    /* hipDeviceSynchronize(); */
    /* cudaCheckErrors("Exec Error"); */
    SWAP(currentGridDevice, nextGridDevice);
  }
  // Copy the final grid back to the host memory.
  /* hipMemcpy2D(finalGameGrid, GhostN * sizeof(bool), currentGridDevice, pitchStart, GhostN * sizeof(bool), */
      /* GhostN, hipMemcpyDeviceToHost); */
  /* cudaCheckErrors("Final Memcpy 2D Error"); */
  hipMemcpy(finalGameGrid, currentGridDevice, GhostN *GhostN * sizeof(bool), hipMemcpyDeviceToHost);


  hipEventRecord(endTimeDevice, 0);
  hipEventSynchronize(endTimeDevice);

  float time;
  hipEventElapsedTime(&time, startTimeDevice, endTimeDevice);
  std::cout << std::endl << prefix << "Execution Time is = <"
    << time / 1000.0f << "> seconds" << std::endl;

  utilities::countGhost(finalGameGrid, N, N, prefix);
  /* utilities::print(finalGameGrid, N + 2); */
  hipFree(currentGridDevice);
  hipFree(nextGridDevice);
  hipDeviceReset();

  delete[] finalGameGrid;
  return;
}

__global__ void simpleNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int index = row * N + col;
  if (col < N && row < N)
  {
    int x = index % N;
    int y = (index - x) / N;
    size_t up = ((y + N - 1) % N) * N;
    size_t center = y * N;
    size_t down = ((y + 1) % N) * N;
    size_t left = (x + N - 1) % N;
    size_t right = (x + 1) % N;

    int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
    nextGrid[center + x] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;
  }
  return;
}

__global__ void simpleNextGenerationKernelPitch(bool* currentGrid, bool* nextGrid, int N,
    size_t currentGridPitch, size_t nextGridPitch)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < N && row < N)
  {
    bool* currentRow = (bool*)(currentGrid + row * currentGridPitch);

    // The row above the current one.
    size_t up = (row + N - 1) % N;
    bool* previousRow = (bool*)(currentGrid + up * currentGridPitch);
    // The row below the current one.
    size_t down = (row + 1) % N;
    bool* nextRow = (bool*)(currentGrid + down * currentGridPitch);
    // Get the index for the left column
    size_t left = (col + N - 1) % N;
    // Get the index of the right column
    size_t right = (col + 1) % N;


    int livingNeighbors = previousRow[left] + previousRow[col] + previousRow[right]
      + currentRow[left] + currentRow[right] + nextRow[left] + nextRow[col] + nextRow[right];

    bool* nextGridRow = (bool*)(nextGrid + row * nextGridPitch);
    nextGridRow[col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentRow[col]) ? 1 : 0;
  }
  return;
}

__global__ void ghostRows(bool* Grid, int N)//Does not  copy corners twp
{
  int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (x < N - 1)
  {
    //The first and last columns are to be wrriten
    Grid[toLinearIndex(N - 1, x, N)] = Grid[toLinearIndex(1, x, N)];  //write bottom to top
    Grid[toLinearIndex(0, x, N)] = Grid[toLinearIndex(N - 2, x, N)];  //write top to bottom
  }
}
__global__ void ghostCols(bool* Grid,int N)//Does not copy corners
{
  int y = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (y< N-1)
  {
    //std:cout<<id;
    Grid[toLinearIndex(y, N - 1, N)] = Grid[toLinearIndex(y, 1, N)];  //write left  to   right
    Grid[toLinearIndex(y, 0, N)] = Grid[toLinearIndex(y, N - 2, N)];  //write right  to left

  }
}

__global__ void ghostCorners(bool* grid, int N)
{
  grid[toLinearIndex(0, 0, N)] = grid[toLinearIndex(N-2, N - 2, N)];//(0,0)-->(N-2,N-2)
  grid[toLinearIndex(N-1, N - 1, N)] = grid[toLinearIndex(1, 1, N)];//(N-1,N-1)-->(1,1)
  grid[toLinearIndex(0, N - 1, N)] = grid[toLinearIndex(N - 2, 1, N)];//(0,N-1)-->(N-2,1)
  grid[toLinearIndex(N - 1, 0, N)] = grid[toLinearIndex(1, N - 2, N)];//(N-1,0)-->(1,N-2)
}


__global__ void simpleGhostNextGenerationKernel(bool* currentGrid, bool* nextGrid, int N)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
  /* int index = row * (N + 2) + col; */
  if ((col < N + 1) && (row < N + 1))
  {
    int x = col;
    int y = row;
    size_t up = ((y - 1)) * (N+2);
    size_t center = y * (N+2);
    size_t down = ((y + 1) ) * (N+2);
    size_t left = (x - 1);
    size_t right = (x + 1);

    int livingNeighbors = calcNeighborsKernel(currentGrid, x, left, right, center, up, down);
    nextGrid[center + x] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentGrid[x + center]) ? 1 : 0;
  }
  return;
}

__global__ void simpleGhostNextGenerationKernelPitch(bool* currentGrid, bool* nextGrid, int N,
    size_t currentGridPitch, size_t nextGridPitch)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
  if ((col < N + 1) && (row < N + 1))
  {
    bool* currentRow = (bool*)(currentGrid + row * currentGridPitch);

    /* The row above the current one. */
    size_t up = (row - 1);
    bool* previousRow = (bool*)(currentGrid + up * currentGridPitch);
    /* The row below the current one. */
    size_t down = (row + 1);
    bool* nextRow = (bool*)(currentGrid + down * currentGridPitch);
    /* Get the index for the left column */
    size_t left = (col - 1);
    /* Get the index of the right column */
    size_t right = (col + 1) ;


    int livingNeighbors = previousRow[left] + previousRow[col] + previousRow[right]
      + currentRow[left] + currentRow[right] + nextRow[left] + nextRow[col] + nextRow[right];

    bool* nextGridRow = (bool*)(nextGrid + row * nextGridPitch);
    nextGridRow[col] = livingNeighbors == 3 ||
      (livingNeighbors == 2 && currentRow[col]) ? 1 : 0;
  }
  return;
}


__device__ int calcNeighborsKernel(bool* currentGrid, int x, int left, int right, int center,
    int up, int down)
{
  return currentGrid[left + up] + currentGrid[x + up]
      + currentGrid[right + up] + currentGrid[left + center]
      + currentGrid[right + center] + currentGrid[left + down]
      + currentGrid[x + down] + currentGrid[right + down];
}

